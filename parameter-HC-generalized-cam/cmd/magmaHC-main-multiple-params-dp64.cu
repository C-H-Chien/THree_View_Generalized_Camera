#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <fstream>
#include <iomanip>
#include <chrono>
#include <string>
// =======================================================================================================
// main function
//
// Modifications
//    Chien  23-02-26    Read Multiple Target Parameter Files and Solve (i) 3-Views With 
//                       4-Points Problem and (ii) Six Lines 6x6 Problem.
//    Chien  24-03-02    For 3-views 4-points problem, only 583 solutions are needed to find 
//                       the actual solution of the problem. This is the same as stated in the paper.
//                       The change is to remove reading 3072 solutions and read 583 solutions.
//
//> (c) LEMS, Brown University
//> Chiang-Heng Chien (chiang-heng_chien@brown.edu)
// =======================================================================================================
//> nvidia cuda
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

//> magma
#include "magma_v2.h"

//> magma
#include "magmaHC-DP64/magmaHC-problems.cuh"

//> p2c
#include "magmaHC-DP64/const-matrices/p2c-symbolic_3views_4pts.h"

//> global repo directory
std::string repo_dir = "/users/cchien3/data/cchien3/bitbucket-repos/gpu-phc/parameter-HC-generalized-cam/";

int main(int argc, char **argv) {
 argc; ++argv;
  std::string arg;
  int argIndx = 0;
  int argTotal = 4;
  std::string HC_problem = "default";

  if (argc) {
    arg = std::string(*argv);
    if (arg == "-h" || arg == "--help") {
      magmaHCWrapperDP64::print_usage();
      exit(1);
    }
    else if (argc <= argTotal) {
      while(argIndx <= argTotal-1) {
        if (arg == "-p" || arg == "--problem") {
          argv++;
          arg = std::string(*argv);
          HC_problem = arg;
          argIndx+=2;
          break;
        }
        else {
          std::cerr<<"invalid input arguments! See examples: \n";
          magmaHCWrapperDP64::print_usage();
          exit(1);
        }
        argv++;
      }
    }
    else if (argc > argTotal) {
      std::cerr<<"too many arguments!\n";
      magmaHCWrapperDP64::print_usage();
      exit(1);
    }
  }
  else { magmaHCWrapperDP64::print_usage(); exit(1); }

  magmaDoubleComplex *h_startSols;
  magmaDoubleComplex *h_Track;
  magmaDoubleComplex *h_startParams;
  magmaDoubleComplex *h_targetParams;
  magmaDoubleComplex *h_phc_coeffs_Hx;
  magmaDoubleComplex *h_phc_coeffs_Ht;
  magma_int_t *h_Hx_idx;
  magma_int_t *h_Ht_idx;

  //> files to be read
  std::string repo_root_dir = repo_dir;
  repo_dir.append("problems/");
  std::string problem_filename = repo_dir.append(HC_problem);

  //> declare class objects (put the long lasting object in dynamic memory)
  magmaHCWrapperDP64::problem_params* pp = new magmaHCWrapperDP64::problem_params;
  magmaHCWrapperDP64::const_mats* cm = new magmaHCWrapperDP64::const_mats;

  pp->define_problem_params(problem_filename, HC_problem);

  //> allocate tracks and coeffs arrays in cpu
  magma_zmalloc_cpu( &h_startSols, pp->numOfTracks*(pp->numOfVars+1) );
  magma_zmalloc_cpu( &h_Track, pp->numOfTracks*(pp->numOfVars+1) );
  magma_zmalloc_cpu( &h_startParams, pp->numOfParams );
  magma_zmalloc_cpu( &h_targetParams, pp->numOfParams );

  magma_zmalloc_cpu( &h_phc_coeffs_Hx, (pp->numOfCoeffsFromParams+1)*(pp->max_orderOf_t+1) );
  magma_zmalloc_cpu( &h_phc_coeffs_Ht, (pp->numOfCoeffsFromParams+1)*(pp->max_orderOf_t) );
  magma_imalloc_cpu( &h_Hx_idx, pp->numOfVars*pp->numOfVars*pp->Hx_maximal_terms*pp->Hx_maximal_parts );
  magma_imalloc_cpu( &h_Ht_idx, pp->numOfVars*pp->Ht_maximal_terms*pp->Ht_maximal_parts );

  std::string startParam_fileName = problem_filename;
  std::string startSols_filename_test = problem_filename;
  if (HC_problem == "3views_4pts")        { 
    startSols_filename_test.append("/start_sols.txt");
    startParam_fileName.append("/start_params.txt");
  }
  else if (HC_problem == "six_lines_6x6") { 
    startSols_filename_test.append("/start_sols.txt");
    startParam_fileName.append("/start_params.txt");
  }
  
  
  std::fstream startCoef_file;
  std::fstream startSols_file;
  bool read_success = 0;
  bool start_sols_read_success = 0;
  bool start_coeffs_read_success = 0;
  
  double s_real, s_imag;
  int d = 0, i = 0; 
  startSols_file.open(startSols_filename_test, std::ios_base::in);
  if (!startSols_file) { std::cerr << "Problem start solutions file NOT existed!\n"; exit(1); }
  else {
    while (startSols_file >> s_real >> s_imag) {
      (h_startSols + i * (pp->numOfVars+1))[d] = MAGMA_Z_MAKE(s_real, s_imag);
      (h_Track + i * (pp->numOfVars+1))[d] = MAGMA_Z_MAKE(s_real, s_imag);
      if (d < pp->numOfVars-1) { d++; }
      else {
        d = 0;
        i++;
      }
    }
    for(int k = 0; k < pp->numOfTracks; k++) {
      (h_startSols + k * (pp->numOfVars+1))[pp->numOfVars] = MAGMA_Z_MAKE(1.0, 0.0);
      (h_Track + k * (pp->numOfVars+1))[pp->numOfVars] = MAGMA_Z_MAKE(1.0, 0.0);
    }
    start_sols_read_success = 1;
  }

  //> read start system parameters
  d = 0;
  startCoef_file.open(startParam_fileName, std::ios_base::in);
  if (!startCoef_file) { std::cerr << "Problem start parameters file NOT existed!\n"; exit(1); }
  else {
    while (startCoef_file >> s_real >> s_imag) {
      (h_startParams)[d] = MAGMA_Z_MAKE(s_real, s_imag);
      d++;
    }
    start_coeffs_read_success = 1;
  }

  //>-------------------------------------------------------------------------------------------------
  bool Hx_file_read_success = false;
  bool Ht_file_read_success = false;
  
  std::string filename_Hx = problem_filename;
  std::string filename_Ht = problem_filename;
  filename_Hx.append("/Hx_idx.txt");
  filename_Ht.append("/Ht_idx.txt");
  std::fstream Hx_idx_file;
  std::fstream Ht_idx_file;
  
  //> 4) read Hx index matrix, if required
  int index;
  d = 0;
  Hx_idx_file.open(filename_Hx, std::ios_base::in);
  if (!Hx_idx_file) { std::cerr << "problem Hx index matrix file not existed!\n"; exit(1); }
  else {
    while (Hx_idx_file >> index) {
      (h_Hx_idx)[d] = index;
      d++;
    }
    Hx_file_read_success = 1;
  }
  //> 5) read Ht index matrix
  d = 0;
  Ht_idx_file.open(filename_Ht, std::ios_base::in);
  if (!Ht_idx_file) { std::cerr << "problem Ht index matrix file not existed!\n"; exit(1);}
  else {
    while (Ht_idx_file >> index) {
      (h_Ht_idx)[d] = index;
      d++;
    }
    Ht_file_read_success = 1;
  }

  //> Write the timings to a file
  std::ofstream timings_file;
  std::string write_timings_file_dir = repo_root_dir;
  write_timings_file_dir.append("Timings_Collection.txt");
  timings_file.open(write_timings_file_dir);
  if ( !timings_file.is_open() ) { std::cout<<"Collection of Timings write files cannot be opened!"<<std::endl; exit(1); }
  
  // =============================================================================
  //> read file: target parameters. Read them iteractively
  // =============================================================================
  bool RANSAC = true;
  int ratio = 90;
  int numOfTargetFiles = 250;
  problem_filename.append("/synthetic_evaluation/");
  for (int tp = 0; tp < numOfTargetFiles; tp++) {

    std::string targetParam_fileName;
    if (RANSAC) { 
      targetParam_fileName = std::string("/users/cchien3/data/cchien3/bitbucket-repos/gpu-phc/auto-gen-tools/RANSAC/outlier_ratio_");
      targetParam_fileName.append(std::to_string(ratio));
      targetParam_fileName.append("/target_params/target_params_"); 
    }
    else { targetParam_fileName = problem_filename; targetParam_fileName.append("/target_params_synthetic/target_params_"); }
    
    //targetParam_fileName.append("target_params_synthetic/target_params_");
    std::string fileIndx = std::to_string(tp);
    std::string extension = ".txt";
    std::string padded_fileName = std::string(6 - std::min(6, (int)(fileIndx.length())), '0') + fileIndx + extension;
    targetParam_fileName.append(padded_fileName);
    std::fstream targetParams_file;
    bool targetParams_read_success = false;

    d = 0;
    targetParams_file.open(targetParam_fileName, std::ios_base::in);
    if (!targetParams_file) { std::cerr << "problem target parameters file not existed!\n"; std::cout << targetParam_fileName << std::endl; exit(1); }
    else {
      while (targetParams_file >> s_real >> s_imag) {
        (h_targetParams)[d] = MAGMA_Z_MAKE(s_real, s_imag);
        d++;
      }
      targetParams_read_success = true;
    }

    //> PHC
    if (HC_problem == "3views_4pts")        { magmaHCWrapperDP64::p2c_symbolic_3views_4pts(h_startParams, h_targetParams, h_phc_coeffs_Hx, h_phc_coeffs_Ht); }
    else if (HC_problem == "six_lines_6x6") { 
      std::fstream phc_coeffs_Hx_file;
      std::fstream phc_coeffs_Ht_file;
      std::string phcHxCoeffs_fileName = problem_filename;
      std::string phcHtCoeffs_fileName = problem_filename;
      phcHxCoeffs_fileName.append("/Hx_numerical_data/Hx_");
      phcHtCoeffs_fileName.append("/Ht_numerical_data/Ht_");
      phcHxCoeffs_fileName.append(padded_fileName);
      phcHtCoeffs_fileName.append(padded_fileName);
      phc_coeffs_Hx_file.open(phcHxCoeffs_fileName, std::ios_base::in);
      phc_coeffs_Ht_file.open(phcHtCoeffs_fileName, std::ios_base::in);
      d = 0;
      //> Hx
      if (!phc_coeffs_Hx_file) { std::cerr << "Numerical PHC Hx file not found!\n"; exit(1); }
      else {
        while (phc_coeffs_Hx_file >> s_real >> s_imag) {
          (h_phc_coeffs_Hx)[d] = MAGMA_Z_MAKE(s_real, s_imag);
          d++;
        }
      }
      //> Ht
      d = 0;
      if (!phc_coeffs_Ht_file) { std::cerr << "Numerical PHC Ht file not found!\n"; exit(1); }
      else {
        while (phc_coeffs_Ht_file >> s_real >> s_imag) {
          (h_phc_coeffs_Ht)[d] = MAGMA_Z_MAKE(s_real, s_imag);
          d++;
        }
      }
    }

    read_success = (start_sols_read_success && start_coeffs_read_success && targetParams_read_success && Hx_file_read_success && Ht_file_read_success);

    //> write only the real solutions to files
    std::ofstream real_sols_file;
    std::string write_real_sols_file_dir = repo_root_dir;
    if (HC_problem == "3views_4pts")        { 
      if (RANSAC) {
        write_real_sols_file_dir = std::string("/users/cchien3/data/cchien3/bitbucket-repos/gpu-phc/auto-gen-tools/RANSAC/outlier_ratio_"); 
        write_real_sols_file_dir.append(std::to_string(ratio));
        write_real_sols_file_dir.append("/GPUHC_Solutions/"); 
      }
      else { write_real_sols_file_dir.append("problems/3views_4pts/synthetic_evaluation/GPUHC_Solutions_DP/"); }
    }
    else if (HC_problem == "six_lines_6x6") { write_real_sols_file_dir.append("problems/six_lines_6x6/synthetic_evaluation/GPUHC_Solutions_DP/"); }
    write_real_sols_file_dir.append(padded_fileName);
    real_sols_file.open(write_real_sols_file_dir);
    if ( !real_sols_file.is_open() ) { std::cout<<"Solutions for evaluation write files cannot be opened!"<<std::endl; exit(1); }

    //> Call homotopy continuation solver
    if (read_success) {
      magmaHCWrapperDP64::homotopy_continuation_solver(h_startSols, h_Track, h_startParams, h_targetParams, h_Hx_idx, h_Ht_idx, 
                                                   h_phc_coeffs_Hx, h_phc_coeffs_Ht, pp, HC_problem, real_sols_file, timings_file
                                                   );
    }
    else {
      std::cout<<"read files failed!"<<std::endl;
      exit(1);
    }

    real_sols_file.close();
  }

  delete pp;
  delete cm;
  magma_free_cpu( h_startSols );
  magma_free_cpu( h_Track );
  magma_free_cpu( h_startParams );
  magma_free_cpu( h_targetParams );
  magma_free_cpu( h_phc_coeffs_Hx );
  magma_free_cpu( h_phc_coeffs_Ht );

  magma_free_cpu( h_Hx_idx );
  magma_free_cpu( h_Ht_idx );

  timings_file.close();

  return 0;
}
